
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512

__global__ void reduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/
    __shared__ float partialSum[2*BLOCK_SIZE];
    unsigned int t = threadIdx.x;
    unsigned int start = 2*blockIdx.x*blockDim.x;
    partialSum[t] = in[start + t];
    partialSum[blockDim.x + t] = in[start + blockDim.x + t];
    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2)
    {
      __syncthreads();
      if (t%stride == 0 && 2*t + start + stride < size)
        partialSum[2*t]+=partialSum[2*t+stride];
    }
    // INSERT KERNEL CODE HERE
    if (t == 0)
      out[blockIdx.x] = partialSum[0];
}



